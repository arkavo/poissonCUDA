#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>
#include <chrono>
#include <string.h>
#include <fstream>
#include <iostream>

using namespace std;

#define RANGE 512
//Global Host data
float DATA[RANGE][RANGE][RANGE];

class Vector3
{
    public:
        float x;
        float y;
        float z;

        void set(float x0,float y0, float z0)
        {
            x = x0;
            y = y0;
            z = z0;
        }
        void display()
        {
            cout << "("<<x<<","<<y<<","<<z<<")\n";
        }
};

class GPU_DATASET
{
    public:
        float *GPU_ADDRESS;
        float *HOST_ADDRESS;
        Vector3 DIMENSIONS;
        Vector3 ST_PT;
        int ACTIVE_DEVICE;

        void SET_DEVICE(int nD)
        {
            ACTIVE_DEVICE = nD;
        }

        void SET_GPU_PTR()
        {
            int xDim = (int)DIMENSIONS.x;
            int yDim = (int)DIMENSIONS.y;
            int zDim = (int)DIMENSIONS.z;
            hipMalloc(&GPU_ADDRESS,sizeof(float)*xDim*yDim*zDim);
            for(int i=ST_PT.x;i<ST_PT.x+DIMENSIONS.x;i++)
            {
                for(int j=ST_PT.y;j<ST_PT.y+DIMENSIONS.y;j++)
                {
                    for(int k=ST_PT.z;k<ST_PT.z+DIMENSIONS.z;k++)
                    {
                       *(GPU_ADDRESS + i + j*xDim + k*yDim*xDim) = *HOST_ADDRESS;
                    }
                }
            }
        }

        void derv_x()
        {

        }
};


__device__ int getGlobalIdx_3D_3D()
{
    int blockId = blockIdx.x+ blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
    int threadId= blockId * (blockDim.x * blockDim.y * blockDim.z)+ (threadIdx.z * (blockDim.x * blockDim.y))+ (threadIdx.y * blockDim.x)+ threadIdx.x;
    return threadId;
}

void printDevProp(hipDeviceProp_t devProp)
{   
    printf("%s\n", devProp.name);
    printf("Major revision number:                   %d\n", devProp.major);
    printf("Minor revision number:                   %d\n", devProp.minor);
    printf("Total global memory:                     %zu", devProp.totalGlobalMem);
    printf("bytes\n");
    printf("Number of multiprocessors:               %d\n", devProp.multiProcessorCount);
    printf("Total amount of shared memory per block: %zu\n",devProp.sharedMemPerBlock);
    printf("Total registers per block:               %d\n", devProp.regsPerBlock);
    printf("Warp size:                               %d\n", devProp.warpSize);
    printf("Maximum memory pitch:                    %zu\n", devProp.memPitch);
    printf("Total amount of constant memory:         %zu\n",   devProp.totalConstMem);
}

void value(int mode, float val)
{
    srand(time(0));
    for(int i=0;i<RANGE;i++)
    {
        for(int j=0;j<RANGE;j++)
        {
            for(int k=0;k<RANGE;k++)
            {
                DATA[i][j][k] = val*(1-mode) + mode*((rand())%100);
            }
        }
    }
}


int main()
{
    //get Devices
    int nDevices;
    /*Hard Set n
    int nDevices = 1;
    */
    hipGetDeviceCount(&nDevices);
    //Device list and properties
    for (int i = 0; i < nDevices; i++) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDevProp(prop);
        printf("Device Number: %d\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %f\n\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
    //end Device list
    //return 0;
    void* ADDRESS_LIST;
    ADDRESS_LIST = (void*)malloc(sizeof(int)*nDevices*nDevices*nDevices);
    ADDRESS_LIST = DATA;
    Vector3 test;
    test.set(RANGE,RANGE,RANGE);
    test.display();
    printf("%p\n",ADDRESS_LIST);
}