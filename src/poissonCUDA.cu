//Three Dimensional Poisson solver
//Author: Arkavo Hait, 2021



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <limits.h>

using namespace std;

//Define Global variables

//#define X 5
//#define Y 5
//#define Z 5

//Box struct
struct BOX
{
    int X;
    int Y;
    int Z;
};
/*
//X derivative (unused)

__global__ void DX(double* R, double* C,double dx)
{
    int index = threadIdx.x*X + blockIdx.x*X*Y +X+X*Y;
    for(int i=1;i<X-1;i++)
    {
        *(R+index+i) = (*(C+index+1+i) - *(C+index-1+i))/(dx*2);
    }
}
//Y derivative (unused)
__global__ void DY(double* R, double* C,double dy)
{
    int index = threadIdx.x + blockIdx.x*X*Y+ X + X*Y;
    for(int i=1;i<Y-1;i++)
    {
        *(R+index+i) = (*(C+index+X*(i+1)) - *(C+index+X*(i-1)))/(dy*2);
    }
}
//Z derivative (unused)
__global__ void DZ(double* R, double* C,double dz)
{
    int index = threadIdx.x + blockIdx.x*X+X+X*Y;
    for(int i=1;i<Z-1;i++)
    {
        *(R+index+i) = (*(C+index+X*Y*(i+1))-*(C+index+X*Y*(i-1)))/(dz*2);
    }
}
*/
//X double derivative
__global__ void DDX(double* R, double* C,int X,int Y,int Z,double dx)
{
    int index = threadIdx.x*X + blockIdx.x*X*Y + X+X*Y;
    for(int i=1;i<X-1;i++)
    {
        *(R+index+i) = (*(C+index+1+i) + *(C+index-1+i) - 2* *(C+index+i))/(dx*dx);
    }
}

//Y double derivative
__global__ void DDY(double* R, double* C,int X,int Y,int Z, double dy)
{
    int index = threadIdx.x + blockIdx.x*X*Y+X+X*Y;
    for(int i=1;i<Y-1;i++)
    {
        *(R+index+i*X) = (*(C+index+X*(i+1)) + *(C+index+X*(i-1)) - 2* *(C+index+i*X))/(dy*dy);
    }
}

//Z double derivative
__global__ void DDZ(double* R, double* C,int X,int Y,int Z, double dz)
{
    int index = threadIdx.x + blockIdx.x*X+X+X*Y;
    for(int i=0;i<Z-1;i++)
    {
        *(R+index+i*X*Y) = (*(C+index+X*Y*(i+1)) + *(C+index+X*Y*(i-1))- 2* *(C+index+i*X*Y))/(dz*dz);
    }
}

//parallel function to update matrices
__global__ void ASSIGN(double* R, double* C,int X,int Y,int Z)
{
    int index = threadIdx.x*X + blockIdx.x*X*Y + X + X*Y ;
    for(int i=1;i<X-1;i++)
    {
        *(R+index+i) = *(C+index+i);
    }
}

//parallel function to add two matrices
__global__ void ADD(double* R,double* C,double dt,int X,int Y,int Z)
{
    int index = threadIdx.x*X+blockIdx.x*X*Y+X+X*Y;
    for(int i=0;i<X;i++)
    {
        *(R+index+i) += (*(C+index+i) * dt);
    }
}

//parallel function to compare two matrices, outputting a maximum difference bteween elements
__global__ void COMPARE(double* R, double* C, double* OUT_H,int X,int Y,int Z)
{
    int index = threadIdx.x * X + blockIdx.x * X * Y+X+X*Y;
    for(int i=1;i<X-1;i++)
    {
        if(abs(*(R+index+i)-*(C+index+i))>= *OUT_H)
        {
            *OUT_H = abs(*(R+index+i) - *(C+index+i));
        }
    }
}

//reset function, use it to reset any pointer
__global__ void RESET_CTR(double* C)
{
    *C = 0;
}

//display function, use to disaply any array
void display(double* DATA,int X,int Y,int Z)
{
    for(int k=0;k<Z;k++)
        {for(int j=0;j<Y;j++)
            {for(int i=0;i<X;i++)
                {printf("%.2lf ",*(DATA+i+X*j+X*Z*k));}
            printf("\n");}printf("\n");}
}

//main fxn, will fix with args after
int main()
{
    //DECLARE YOUR VARIABLES HERE
    
    struct BOX grid = {30,30,30};
    //step for double derivatives
    double step = 0.01;
    //tolerence
    double tol = 0.00001;
    
    const int X = grid.X;
    const int Y = grid.Y;
    const int Z = grid.Z;
    printf("\n%d %d %d\n",X,Y,Z);
    unsigned long long SIZE_0 = ((int)sizeof(double))*X*Y*Z;
    double* DATA_H; 
    double* DATA_F;
    DATA_H = (double*)malloc(SIZE_0);
    DATA_F = (double*)malloc(SIZE_0);
    
    //Impose Boundary conditions here
    
    for(int i=0;i<X;i++)
        {for(int j=0;j<Y;j++)
            {for(int k=0;k<Z;k++)
                {
                    if(i==0||i==(X-1)||j==0||j==(Y-1)||k==0||k==(Z-1))
                        *(DATA_H+i+j*X+k*X*Y) = 5;
                    else
                        *(DATA_H+i+j*X+k*X*Y) = 0;
    }}}
    
    
    //error counters
    double* CC;
    double CCD = 0;
    
    //DATA pointer for device state 0
    double* DATA_ORIGINAL;
    //DATA pointer for device state 1
    double* DATA_NEXT;
    //Derivative results pointers
    double* DDX_D;
    double* DDY_D;
    double* DDZ_D;
    
    //array size for device
    
    //allocating space for arrays
    hipMalloc((void**)&DATA_ORIGINAL,SIZE_0);
    
    hipMalloc((void**)&DATA_NEXT,SIZE_0);
    
    hipMalloc((void**)&DDX_D,SIZE_0);
    hipMalloc((void**)&DDY_D,SIZE_0);
    hipMalloc((void**)&DDZ_D,SIZE_0);
    
    hipMalloc(&CC,(int)sizeof(double));
    
    //token counter
    CCD = 10;
    //copy data state0, state1 
    hipMemcpy(DATA_ORIGINAL,DATA_H,SIZE_0,hipMemcpyHostToDevice);
    hipMemcpy(DATA_NEXT,DATA_H,SIZE_0,hipMemcpyHostToDevice);
    //data arrays for final copies
    //double DATA_F[X][Y][Z];
    
    //double DATA_FF[X][Y][Z];
    
    //cudaMemcpy(&DATA_FF,DATA_ORIGINAL,SIZE_0,cudaMemcpyDeviceToHost);
    
    //counter
    int ct = 0;
    
    //run while tolerence > difference
    while(CCD>tol)
    {
        //reset difference every loop
        RESET_CTR  <<<1,1>>>  (CC);
        //run derivatives
        DDY <<<Z-2,X-2>>> (DDY_D,DATA_ORIGINAL,X,Y,Z,10.);
        DDZ <<<X-2,Y-2>>> (DDZ_D,DATA_ORIGINAL,X,Y,Z,10.);
        DDX <<<Y-2,Z-2>>> (DDX_D,DATA_ORIGINAL,X,Y,Z,10.);     //add into state 1
        ADD <<<Y-2,Z-2>>> (DATA_NEXT,DDX_D,step,X,Y,Z);
        ADD <<<Y-2,Z-2>>> (DATA_NEXT,DDY_D,step,X,Y,Z);
        ADD <<<Y-2,Z-2>>> (DATA_NEXT,DDZ_D,step,X,Y,Z);
        //compare state1 state 0
        COMPARE<<<Y-2,Z-2>>>(DATA_ORIGINAL,DATA_NEXT,CC,X,Y,Z);
        //copy back max error
        hipMemcpy(&CCD,CC,sizeof(double),hipMemcpyDeviceToHost);
        //make state 1 as state0
        ASSIGN  <<<Y-2,Z-2>>>  (DATA_ORIGINAL,DATA_NEXT,X,Y,Z);
        //update counter
        ct += 1;
        //information every 1000 loops because of visibility
        if(ct%1000==0)
        {
            hipMemcpy(&CCD,CC,sizeof(double),hipMemcpyDeviceToHost);
            printf("%d loops %0.6lf max error\r",ct,CCD);
        }
    }
    //copy back final array
    hipMemcpy(DATA_F,DATA_ORIGINAL,SIZE_0,hipMemcpyDeviceToHost);
    //final print statement
    printf("\n\nConverged in %d loops\n\n",ct-1);
    //display optional
    //display(DATA_F,X,Y,Z);
    //free pointers
    hipFree(DATA_ORIGINAL);
    hipFree(DATA_NEXT);
    hipFree(DDX_D);
    hipFree(DDY_D);
    hipFree(DDZ_D);
    
    return 0;
}