#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>
#include <chrono>
#include <string.h>
#include <fstream>
#include <iostream>
//using namespace std;
#define RANGE 4
#define ITER 10000
#define tol 0.001
#define dr 1
#define dt 0.2
//ALL HOST TESTING COPIES COPIUM
float field[RANGE][RANGE][RANGE];
float temp_field[RANGE][RANGE][RANGE];
//First Derivatives
float dx[RANGE][RANGE][RANGE];
float dy[RANGE][RANGE][RANGE];
float dz[RANGE][RANGE][RANGE];
//Second Derivatives
float ddx[RANGE][RANGE][RANGE];
float ddy[RANGE][RANGE][RANGE];
float ddz[RANGE][RANGE][RANGE];
//DEVICE COPIES TO BE ALLOCATED DYNAMICALLY

int FX = 0;

void printDevProp(hipDeviceProp_t devProp)
{   
    printf("%s\n", devProp.name);
    printf("Major revision number:                   %d\n", devProp.major);
    printf("Minor revision number:                   %d\n", devProp.minor);
    printf("Total global memory:                     %zu", devProp.totalGlobalMem);
    printf("bytes\n");
    printf("Number of multiprocessors:               %d\n", devProp.multiProcessorCount);
    printf("Total amount of shared memory per block: %zu\n",devProp.sharedMemPerBlock);
    printf("Total registers per block:               %d\n", devProp.regsPerBlock);
    printf("Warp size:                               %d\n", devProp.warpSize);
    printf("Maximum memory pitch:                    %zu\n", devProp.memPitch);
    printf("Total amount of constant memory:         %zu\n", devProp.totalConstMem);
}
void device_list()
{
    //get Devices
    int nDevices;
    /*Hard Set n
    int nDevices = 1;
    */
    hipGetDeviceCount(&nDevices);
    //Device list and properties
    for (int i = 0; i < nDevices; i++) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDevProp(prop);
        printf("Device Number: %d\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %f\n\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

void display()
{
    for(int i=0;i<RANGE;i++)
    {
        for(int j=0;j<RANGE;j++)
        {
            for(int k=0;k<RANGE;k++)
            {
                std::cout<<field[i][j][k]<<" ";
            }
            std::cout<<"\n";
        }
        std::cout<<"\n\n";
    }
}

void deriv(int mode)
{
    if(mode==1)
    {
        for(int i=1;i<RANGE-1;i++)
        {
            for(int j=1;j<RANGE-1;j++)
            {
                for(int k=1;k<RANGE-1;k++)
                {
                    dx[i][j][k] = (field[i+1][j][k] - field[i-1][j][k])/(2.0*dr);
                    dx[0][j][k] = dx[1][j][k];
                    dx[RANGE][j][k] = dx[RANGE-1][j][k];
                    
                    dy[i][j][k] = (field[i][j+1][k] - field[i][j-1][k])/(2.0*dr);
                    dy[i][0][k] = dy[i][1][k];
                    dy[i][RANGE][k] = dy[i][RANGE-1][k];

                    dz[i][j][k] = (field[i][j][k+1] - field[i][j][k-1])/(2.0*dr);
                    dz[i][j][0] = dz[i][j][1];
                    dz[i][j][RANGE-1] = dz[i][j][RANGE];
                }
            }
        }
    }

    if(mode==2)
    {
        for(int i=1;i<RANGE-1;i++)
        {
            for(int j=1;j<RANGE-1;j++)
            {
                for(int k=1;k<RANGE-1;k++)
                {
                    ddx[i][j][k] = (dx[i+1][j][k] - dx[i-1][j][k])/(2.0*dr);
                    ddx[0][j][k] = ddx[1][j][k];
                    ddx[RANGE][j][k] = ddx[RANGE-1][j][k];
                    
                    ddy[i][j][k] = (dy[i][j+1][k] - dy[i][j-1][k])/(2.0*dr);
                    ddy[i][0][k] = ddy[i][1][k];
                    ddy[i][RANGE][k] = ddy[i][RANGE-1][k];

                    ddz[i][j][k] = (dz[i][j][k+1] - dz[i][j][k-1])/(2.0*dr);
                    ddz[i][j][0] = ddz[i][j][1];
                    ddz[i][j][RANGE-1] = ddz[i][j][RANGE];
                }
            }
        }
    }
}

void DECLARE(int nDevice)
{
    unsigned long long PARTITION = (sizeof(double)*(RANGE-1)*(RANGE-1)*(RANGE-1) )/nDevice + 1;
    for(int i=0;i<nDevice;i++)
    {
        hipSetDevice(i);
        double* dx, dy, dz, ddx, ddy, ddz;
        hipMalloc((void**)&dx, PARTITION);
        hipMalloc((void**)&dy, PARTITION);
        hipMalloc((void**)&dz, PARTITION);
        hipMalloc((void**)&ddx, PARTITION);
        hipMalloc((void**)&ddy, PARTITION);
        hipMalloc((void**)&ddz, PARTITION);
    }
}
//SINGLE GPU SINGLE THREAD FORCE METHOD
__global__ void DEVICE_DIFF_FXN(double* result, double* data,double* d,double* dd,int dimension,int order)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(dimension==1 && order==1)
    {
        if(index%(RANGE-1)!=0)
        {
            d[index] = data[index+1] - data[index];
        }
    }
    if(dimension==2 && order==1)
    {
        if(index%((RANGE-1)*RANGE-1)!=0)
        {
            d[index] = data[index+RANGE] - data[index];
        }
    }
    if(dimension==3 && order==1)
    {
        if(index%((RANGE-1)*(RANGE-1)*RANGE-1)!=0)
        {
            d[index] = data[index+RANGE*RANGE] - data[index];
        }
    }
    if(dimension==1 && order==2)
    {
        if(index%(RANGE-1)!=0)
        {
            dd[index] = d[index+1] - d[index];
        }
    }
    if(dimension==2 && order==2)
    {
        if(index%((RANGE-1)*RANGE-1)!=0)
        {
            dd[index] = d[index+RANGE] - d[index];
        }
    }
    if(dimension==3 && order==2)
    {
        if(index%((RANGE-1)*(RANGE-1)*RANGE-1)!=0)
        {
            dd[index] = d[index+RANGE*RANGE] - d[index];
        }
    }
    if(order==1)
        result = d;
    if(order==2)
        result = dd;
}
__global__ void INITIAL(double* DEVICE_DATA)
{
    for(int i=0;i<RANGE;i++)
    {
        for(int j=0;j<RANGE;j++)
        {
            for(int k=0;k<RANGE;k++)
            {
                if(i==0||i==RANGE-1||j==0||j==RANGE-1||k==0||k==RANGE-1)
                    *(DEVICE_DATA+i+j*RANGE+k*RANGE*RANGE) = 20.;
                else
                    *(DEVICE_DATA+i+j*RANGE+k*RANGE*RANGE) = 0.;
            }
        }
    }
}
__host__ double* UPDATE(double* DEVICE_DATA, double* up, int mode)
{
    if (mode==1)
    {
        for(int i=0;i<RANGE;i++)
            for(int j=0;j<RANGE;j++)
                for(int k=0;k<RANGE;k++)
                    *(DEVICE_DATA+i+j*RANGE+k*RANGE*RANGE) += *(up+i+j*RANGE+k*RANGE*RANGE) * dt;
    }
    if (mode==2)
    {
        for(int i=0;i<RANGE;i++)
            for(int j=0;j<RANGE;j++)
                for(int k=0;k<RANGE;k++)
                    *(DEVICE_DATA+i+j*RANGE+k*RANGE*RANGE) += *(up+i+j*RANGE+k*RANGE*RANGE) * dt*dt/2;
    }
    return DEVICE_DATA;
}
__global__ void UPDATE(double* R,double* S)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    for(int i=0;i<RANGE;i++)
    {
        R[index+i] += S[index+i];
        S[index+i] = 0;
    }
}
__global__ void ERROR_CHECK(double* DEVICE_P, double* DEVICE_C)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x; 
    double err_max = 0.01;
    for(int i=0;i<RANGE;i++)
    {
        if((DEVICE_C[index+i]-DEVICE_P[index+1])<=err_max)
            FX = 1;        
    }
}
int poisson3d(double* DEVICE_P, double* DEVICE_C, double* DEVICE_CX,double* DEVICE_CXX, double* DEVICE_S_CX,double* DEVICE_S_CXX)
{
    int loopctr = 0;
    double error = 10000.0;
    double err_max = 0.01;
    while (FX==0)
    {
        loopctr += 1;
        INITIAL<<<1,1>>>(DEVICE_P);
        DEVICE_DIFF_FXN<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_C,DEVICE_CX,DEVICE_CXX,1,1);
        UPDATE<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_CX);
        DEVICE_DIFF_FXN<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_C,DEVICE_CX,DEVICE_CXX,1,2);
        UPDATE<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_CX);
        DEVICE_DIFF_FXN<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_C,DEVICE_CX,DEVICE_CXX,1,3);
        UPDATE<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_CX);
        DEVICE_DIFF_FXN<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_C,DEVICE_CX,DEVICE_CXX,2,1);
        UPDATE<<<RANGE,RANGE>>>(DEVICE_S_CXX,DEVICE_CXX);
        DEVICE_DIFF_FXN<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_C,DEVICE_CX,DEVICE_CXX,2,2);
        UPDATE<<<RANGE,RANGE>>>(DEVICE_S_CXX,DEVICE_CXX);
        DEVICE_DIFF_FXN<<<RANGE,RANGE>>>(DEVICE_S_CX,DEVICE_C,DEVICE_CX,DEVICE_CXX,2,3);
        UPDATE<<<RANGE,RANGE>>>(DEVICE_S_CXX,DEVICE_CXX);
    }
    return loopctr;
}

int main(int argc, char* argv[])
{
    //boundary conditions + init
    for(int i=0;i<RANGE;i++)
    {
        for(int j=0;j<RANGE;j++)
        {
            for(int k=0;k<RANGE;k++)
            {
                if(i==0||i==RANGE-1||j==0||j==RANGE-1||k==0||k==RANGE-1)
                    field[i][j][k] = 20;
                else
                    field[i][j][k] = 0;
            }
        }
    }
    //GPU COPIES
    double* DEVICE_P;
    double* DEVICE_C;
    double* DEVICE_CX;
    double* DEVICE_S_CX;
    double* DEVICE_CXX;
    double* DEVICE_S_CXX;
    double* trial;
    unsigned long long TOTAL_SIZE = sizeof(double)*RANGE*RANGE*RANGE;
    hipMalloc((void**)&DEVICE_P,TOTAL_SIZE);
    hipMalloc((void**)&DEVICE_C,TOTAL_SIZE);
    hipMalloc((void**)&DEVICE_CX,TOTAL_SIZE);
    hipMalloc((void**)&DEVICE_S_CX,TOTAL_SIZE);
    hipMalloc((void**)&DEVICE_CXX,TOTAL_SIZE);
    hipMalloc((void**)&DEVICE_S_CXX,TOTAL_SIZE);
    trial = (double*)malloc(TOTAL_SIZE);
    hipMemcpy(&DEVICE_C,field,TOTAL_SIZE,hipMemcpyHostToDevice);
    
    hipMemcpy(&trial,&DEVICE_C,TOTAL_SIZE,hipMemcpyDeviceToHost);
    
    for(int i=0;i<RANGE;i++){
        for(int j=0;j<RANGE;j++){
            for(int k=0;k<RANGE;k++){
                printf("%f \n",*(trial+i+j*RANGE+j*RANGE*RANGE));}}}
    printf("\n");
    //display();
    
    //Linear cpu time
    
    // float err_max = 10.0;
    // int loopctr = 0;
    // auto hst_st = std::chrono::high_resolution_clock::now();
    // while(abs(err_max)>tol)
    // {
    //     deriv(1);
    //     deriv(2);

        
    //     for(int i=0;i<RANGE;i++)
    //     {
    //         for(int j=0;j<RANGE;j++)
    //         {
    //             for(int k=0;k<RANGE;k++)
    //             {
    //                 dx[i][j][k]+=(ddx[i][j][k])*dt;
    //                 dy[i][j][k]+=(ddy[i][j][k])*dt;
    //                 dz[i][j][k]+=(ddz[i][j][k])*dt;
    //             }
    //         }
    //     }
    //     for(int i=1;i<RANGE-1;i++)
    //     {
    //         for(int j=1;j<RANGE-1;j++)
    //         {
    //             for(int k=1;k<RANGE-1;k++)
    //             {
    //                 temp_field[i][j][k] = field[i][j][k]+(dx[i][j][k]+dy[i][j][k]+dz[i][j][k])*dt;      
    //             }
    //         }
    //     }
    //     for(int i=1;i<RANGE-1;i++)
    //     {
    //         for(int j=1;j<RANGE-1;j++)
    //         {
    //             for(int k=1;k<RANGE-1;k++)
    //             {
    //                 err_max = temp_field[i][j][k] - field[i][j][k];
    //                 loopctr++;
    //                 //if(loopctr%2000000==0)
    //                 //    std::cout<<loopctr<<" iters\n";
    //                 field[i][j][k] = temp_field[i][j][k];
    //             }
    //         }
    //     }
        
    // }
    // auto hst_en = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<float> duration = hst_en-hst_st;
    // std::cout<<"Duration: "<<duration.count()<<"\n";
    //std::cout<<"With "<<loopctr<<" loops\n\n";
    //std::cout<<"Error: "<<err_max<<"\n";
    //device_list();
    //display();
}